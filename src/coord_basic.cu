#include "hip/hip_runtime.h"
#include "coord_basic.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// CUDA kernel for computing distances
__global__ void distcoord_compute_kernel(
    const float* __restrict__ pos1_data,  // Position data for first atom set
    const float* __restrict__ pos2_data,  // Position data for second atom set  
    const int* __restrict__ atom_pairs,   // Atom index pairs [n_elem*2]
    float* __restrict__ output_data,      // Output distances [n_elem]
    float* __restrict__ deriv_data,       // Derivative data [n_elem*3]
    int n_elem,                          // Number of distance calculations
    int stride                           // Stride for position arrays (4 for Float4 alignment)
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n_elem) return;
    
    // Get atom indices for this distance calculation
    int atom1_idx = atom_pairs[idx * 2 + 0];
    int atom2_idx = atom_pairs[idx * 2 + 1];
    
    // Load positions (assuming Float4 storage with stride=4)
    float3 pos1 = make_float3(
        pos1_data[atom1_idx * stride + 0],
        pos1_data[atom1_idx * stride + 1], 
        pos1_data[atom1_idx * stride + 2]
    );
    
    float3 pos2 = make_float3(
        pos2_data[atom2_idx * stride + 0],
        pos2_data[atom2_idx * stride + 1],
        pos2_data[atom2_idx * stride + 2]  
    );
    
    // Compute displacement vector
    float3 disp = make_float3(
        pos1.x - pos2.x,
        pos1.y - pos2.y, 
        pos1.z - pos2.z
    );
    
    // Compute distance
    float dist_sq = disp.x * disp.x + disp.y * disp.y + disp.z * disp.z;
    float dist = sqrtf(dist_sq);
    
    // Store distance
    output_data[idx] = dist;
    
    // Compute and store derivative (unit vector)
    if (dist > 1e-8f) {
        float inv_dist = 1.0f / dist;
        deriv_data[idx * 3 + 0] = disp.x * inv_dist;
        deriv_data[idx * 3 + 1] = disp.y * inv_dist;
        deriv_data[idx * 3 + 2] = disp.z * inv_dist;
    } else {
        deriv_data[idx * 3 + 0] = 0.0f;
        deriv_data[idx * 3 + 1] = 0.0f;
        deriv_data[idx * 3 + 2] = 0.0f;
    }
}

// CUDA kernel for propagating derivatives
__global__ void distcoord_deriv_kernel(
    const int* __restrict__ atom_pairs,    // Atom index pairs [n_elem*2]
    const float* __restrict__ deriv_data,  // Derivative data [n_elem*3]
    const float* __restrict__ sens_data,   // Sensitivity data [n_elem]
    float* __restrict__ pos1_sens,         // Position sensitivity for first atom set
    float* __restrict__ pos2_sens,         // Position sensitivity for second atom set
    int n_elem,                           // Number of distance calculations
    int stride                            // Stride for position arrays
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= n_elem) return;
    
    // Get atom indices for this distance calculation
    int atom1_idx = atom_pairs[idx * 2 + 0];
    int atom2_idx = atom_pairs[idx * 2 + 1];
    
    // Get sensitivity value
    float sens = sens_data[idx];
    
    // Get derivative vector
    float3 deriv = make_float3(
        deriv_data[idx * 3 + 0],
        deriv_data[idx * 3 + 1],
        deriv_data[idx * 3 + 2]
    );
    
    // Scale derivative by sensitivity
    float3 scaled_deriv = make_float3(
        deriv.x * sens,
        deriv.y * sens,
        deriv.z * sens
    );
    
    // Accumulate derivatives (atomic operations for thread safety)
    atomicAdd(&pos1_sens[atom1_idx * stride + 0], scaled_deriv.x);
    atomicAdd(&pos1_sens[atom1_idx * stride + 1], scaled_deriv.y);
    atomicAdd(&pos1_sens[atom1_idx * stride + 2], scaled_deriv.z);
    
    atomicAdd(&pos2_sens[atom2_idx * stride + 0], -scaled_deriv.x);
    atomicAdd(&pos2_sens[atom2_idx * stride + 1], -scaled_deriv.y);
    atomicAdd(&pos2_sens[atom2_idx * stride + 2], -scaled_deriv.z);
}

// Launcher functions
void distcoord_compute_device(
    const float* pos1_data, const float* pos2_data,
    const int* atom_pairs, float* output_data, float* deriv_data,
    int n_elem, int stride, int threadsPerBlock
) {
    int blocksPerGrid = (n_elem + threadsPerBlock - 1) / threadsPerBlock;
    
    distcoord_compute_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        pos1_data, pos2_data, atom_pairs, output_data, deriv_data, n_elem, stride
    );
}

void distcoord_deriv_device(
    const int* atom_pairs, const float* deriv_data, const float* sens_data,
    float* pos1_sens, float* pos2_sens,
    int n_elem, int stride, int threadsPerBlock
) {
    int blocksPerGrid = (n_elem + threadsPerBlock - 1) / threadsPerBlock;
    
    distcoord_deriv_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        atom_pairs, deriv_data, sens_data, pos1_sens, pos2_sens, n_elem, stride
    );
}
