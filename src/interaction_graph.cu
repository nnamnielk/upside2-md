#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

// Simple CUDA error checking macro
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Kernel to check if particles have moved beyond the cache buffer
__global__ void check_cache_validity_kernel(
    int* d_rebuild_flag,
    int n_elem,
    const float* __restrict__ aligned_pos,
    const float* __restrict__ cache_pos,
    const int* __restrict__ id,
    const int* __restrict__ cache_id,
    float max_cache_dist2,
    int pos_stride)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elem) return;

    if (atomicCAS(d_rebuild_flag, 1, 1) == 1) return; // Early exit if another thread already flagged for rebuild

    // Check ID change
    if (id[i] != cache_id[i]) {
        atomicCAS(d_rebuild_flag, 0, 1);
        return;
    }

    // Check position deviation with correct stride
    float dx = aligned_pos[i*pos_stride + 0] - cache_pos[i*4 + 0];
    float dy = aligned_pos[i*pos_stride + 1] - cache_pos[i*4 + 1];
    float dz = aligned_pos[i*pos_stride + 2] - cache_pos[i*4 + 2];
    float dist2 = dx*dx + dy*dy + dz*dz;

    if (dist2 > max_cache_dist2) {
        atomicCAS(d_rebuild_flag, 0, 1);
    }
}

// Kernel to refine cached edges on the GPU
__global__ void find_edges_kernel(
    int* d_n_edge,
    int cache_n_edge,
    int max_n_edge,
    float cutoff2,
    bool symmetric,
    int pos1_stride, int pos2_stride,
    int n_elem1, int n_elem2,
    const int32_t* __restrict__ d_cache_edge_indices1, const int32_t* __restrict__ d_cache_edge_indices2,
    const int32_t* __restrict__ d_cache_edge_id1,      const int32_t* __restrict__ d_cache_edge_id2,
    const float* __restrict__ d_aligned_pos1,    const float* __restrict__ d_aligned_pos2,
    int32_t* __restrict__ d_edge_indices1, int32_t* __restrict__ d_edge_indices2,
    int32_t* __restrict__ d_edge_id1,      int32_t* __restrict__ d_edge_id2)
{
    int i_edge = blockIdx.x * blockDim.x + threadIdx.x;
    if (i_edge >= cache_n_edge) return;

    int32_t i1 = d_cache_edge_indices1[i_edge];
    int32_t i2 = d_cache_edge_indices2[i_edge];

    // Add bounds checking
    if (i1 >= n_elem1 || i2 >= n_elem2 || i1 < 0 || i2 < 0) return;

    const float* p1 = d_aligned_pos1 + pos1_stride * i1;
    const float* p2 = d_aligned_pos2 + pos2_stride * i2;

    float dx = p1[0] - p2[0];
    float dy = p1[1] - p2[1];
    float dz = p1[2] - p2[2];
    float dist2 = dx*dx + dy*dy + dz*dz;

    if (dist2 < cutoff2) {
        int ne = atomicAdd(d_n_edge, 1);
        // Add bounds checking for atomic result
        if (ne >= max_n_edge) return;
        d_edge_indices1[ne] = i1;
        d_edge_indices2[ne] = i2;
        d_edge_id1[ne]      = d_cache_edge_id1[i_edge];
        d_edge_id2[ne]      = d_cache_edge_id2[i_edge];
    }
}


// C-style wrapper functions to be called from interaction_graph.h
extern "C" {

bool ensure_cache_valid_cuda(
    int n_elem1, int n_elem2, bool symmetric,
    const float* aligned_pos1, const int pos1_stride, const int* id1, // Use const for input pointers
    const float* aligned_pos2, const int pos2_stride, const int* id2,
    float cache_cutoff, float cutoff,
    const float* cache_pos1, const float* cache_pos2,
    const int* cache_id1, const int* cache_id2)
{
    int* d_rebuild_flag;
    gpuErrchk(hipMalloc(&d_rebuild_flag, sizeof(int)));
    gpuErrchk(hipMemset(d_rebuild_flag, 0, sizeof(int)));

    // <<< MODIFICATION START >>>
    // Allocate device memory for all kernel inputs
    float* d_aligned_pos1, *d_cache_pos1, *d_aligned_pos2, *d_cache_pos2;
    int *d_id1, *d_cache_id1, *d_id2, *d_cache_id2;

    gpuErrchk(hipMalloc(&d_aligned_pos1, n_elem1 * pos1_stride * sizeof(float)));
    gpuErrchk(hipMalloc(&d_cache_pos1,   n_elem1 * 4 * sizeof(float))); // cache_pos is always stride 4
    gpuErrchk(hipMalloc(&d_id1,          n_elem1 * sizeof(int)));
    gpuErrchk(hipMalloc(&d_cache_id1,    n_elem1 * sizeof(int)));

    // Copy data from host to device
    gpuErrchk(hipMemcpy(d_aligned_pos1, aligned_pos1, n_elem1 * pos1_stride * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cache_pos1,   cache_pos1,   n_elem1 * 4 * sizeof(float),           hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_id1,          id1,          n_elem1 * sizeof(int),                 hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cache_id1,    cache_id1,    n_elem1 * sizeof(int),                 hipMemcpyHostToDevice));
    // <<< MODIFICATION END >>>

    float max_cache_dist2 = 0.25f * (cache_cutoff - cutoff) * (cache_cutoff - cutoff);
    int threadsPerBlock = 256;

    // Check first set of elements
    int blocks1 = (n_elem1 + threadsPerBlock - 1) / threadsPerBlock;
    // <<< MODIFICATION: Use device pointers in kernel launch >>>
    check_cache_validity_kernel<<<blocks1, threadsPerBlock>>>(
        d_rebuild_flag, n_elem1, d_aligned_pos1, d_cache_pos1, d_id1, d_cache_id1, max_cache_dist2, pos1_stride);

    // Check second set if not symmetric
    if (!symmetric) {
        // <<< MODIFICATION START >>>
        // Allocate and copy data for the second set of elements
        gpuErrchk(hipMalloc(&d_aligned_pos2, n_elem2 * pos2_stride * sizeof(float)));
        gpuErrchk(hipMalloc(&d_cache_pos2,   n_elem2 * 4 * sizeof(float)));
        gpuErrchk(hipMalloc(&d_id2,          n_elem2 * sizeof(int)));
        gpuErrchk(hipMalloc(&d_cache_id2,    n_elem2 * sizeof(int)));

        gpuErrchk(hipMemcpy(d_aligned_pos2, aligned_pos2, n_elem2 * pos2_stride * sizeof(float), hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_cache_pos2,   cache_pos2,   n_elem2 * 4 * sizeof(float),           hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_id2,          id2,          n_elem2 * sizeof(int),                 hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_cache_id2,    cache_id2,    n_elem2 * sizeof(int),                 hipMemcpyHostToDevice));
        // <<< MODIFICATION END >>>

        int blocks2 = (n_elem2 + threadsPerBlock - 1) / threadsPerBlock;
        // <<< MODIFICATION: Use device pointers in kernel launch >>>
        check_cache_validity_kernel<<<blocks2, threadsPerBlock>>>(
            d_rebuild_flag, n_elem2, d_aligned_pos2, d_cache_pos2, d_id2, d_cache_id2, max_cache_dist2, pos2_stride);
    }
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    int rebuild_flag = 0;
    gpuErrchk(hipMemcpy(&rebuild_flag, d_rebuild_flag, sizeof(int), hipMemcpyDeviceToHost));
    
    // <<< MODIFICATION START >>>
    // Free all the device memory we allocated
    gpuErrchk(hipFree(d_rebuild_flag));
    gpuErrchk(hipFree(d_aligned_pos1));
    gpuErrchk(hipFree(d_cache_pos1));
    gpuErrchk(hipFree(d_id1));
    gpuErrchk(hipFree(d_cache_id1));
    if (!symmetric) {
        gpuErrchk(hipFree(d_aligned_pos2));
        gpuErrchk(hipFree(d_cache_pos2));
        gpuErrchk(hipFree(d_id2));
        gpuErrchk(hipFree(d_cache_id2));
    }
    // <<< MODIFICATION END >>>
    
    return rebuild_flag == 0; // Return true if cache is still valid
}



void find_edges_cuda(
    int& n_edge, int max_n_edge, int cache_n_edge,
    float cutoff, bool symmetric,
    int n_elem1, int n_elem2, int pos1_stride, int pos2_stride,
    const float* aligned_pos1, const float* aligned_pos2,
    const int32_t* cache_edge_indices1, const int32_t* cache_edge_indices2,
    const int32_t* cache_edge_id1,      const int32_t* cache_edge_id2,
    int32_t* edge_indices1, int32_t* edge_indices2,
    int32_t* edge_id1,      int32_t* edge_id2)
{
    // Validate input parameters
    if (n_elem1 <= 0 || n_elem2 <= 0 || cache_n_edge < 0 || max_n_edge <= 0) {
        return;
    }
    
    if (pos1_stride <= 0 || pos2_stride <= 0) {
        return;
    }
    
    // Check host pointers
    if (!aligned_pos1) { return; }
    if (!symmetric && !aligned_pos2) { return; }
    if (!cache_edge_indices1) { return; }
    if (!cache_edge_indices2) { return; }
    if (!cache_edge_id1) { return; }
    if (!cache_edge_id2) { return; }
    if (!edge_indices1) { return; }
    if (!edge_indices2) { return; }
    if (!edge_id1) { return; }
    if (!edge_id2) { return; }
    
    // Device memory allocation
    float* d_aligned_pos1 = nullptr;
    float* d_aligned_pos2 = nullptr;
    int32_t* d_cache_edge_indices1 = nullptr;
    int32_t* d_cache_edge_indices2 = nullptr;
    int32_t* d_cache_edge_id1 = nullptr;
    int32_t* d_cache_edge_id2 = nullptr;
    int32_t* d_edge_indices1 = nullptr;
    int32_t* d_edge_indices2 = nullptr;
    int32_t* d_edge_id1 = nullptr;
    int32_t* d_edge_id2 = nullptr;
    int* d_n_edge = nullptr;

    gpuErrchk(hipMalloc((void**)&d_aligned_pos1, n_elem1 * pos1_stride * sizeof(float)));
    
    if (!symmetric) {
        gpuErrchk(hipMalloc((void**)&d_aligned_pos2, n_elem2 * pos2_stride * sizeof(float)));
    } else {
        d_aligned_pos2 = d_aligned_pos1;
    }

    gpuErrchk(hipMalloc((void**)&d_cache_edge_indices1, cache_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_cache_edge_indices2, cache_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_cache_edge_id1,      cache_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_cache_edge_id2,      cache_n_edge * sizeof(int32_t)));

    gpuErrchk(hipMalloc((void**)&d_edge_indices1, max_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_edge_indices2, max_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_edge_id1,      max_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_edge_id2,      max_n_edge * sizeof(int32_t)));
    gpuErrchk(hipMalloc((void**)&d_n_edge, sizeof(int)));
    gpuErrchk(hipMemset(d_n_edge, 0, sizeof(int)));

    // HtoD transfers
    gpuErrchk(hipMemcpy(d_aligned_pos1, aligned_pos1, n_elem1 * pos1_stride * sizeof(float), hipMemcpyHostToDevice));
    
    if (!symmetric) {
        gpuErrchk(hipMemcpy(d_aligned_pos2, aligned_pos2, n_elem2 * pos2_stride * sizeof(float), hipMemcpyHostToDevice));
    }
    
    gpuErrchk(hipMemcpy(d_cache_edge_indices1, cache_edge_indices1, cache_n_edge * sizeof(int32_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cache_edge_indices2, cache_edge_indices2, cache_n_edge * sizeof(int32_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cache_edge_id1,      cache_edge_id1,      cache_n_edge * sizeof(int32_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_cache_edge_id2,      cache_edge_id2,      cache_n_edge * sizeof(int32_t), hipMemcpyHostToDevice));

    // Kernel launch
    int threadsPerBlock = 256;
    int blocks = (cache_n_edge + threadsPerBlock - 1) / threadsPerBlock;
    
    find_edges_kernel<<<blocks, threadsPerBlock>>>(
        d_n_edge, cache_n_edge, max_n_edge, cutoff * cutoff, symmetric, pos1_stride, pos2_stride,
        n_elem1, n_elem2,
        d_cache_edge_indices1, d_cache_edge_indices2, d_cache_edge_id1, d_cache_edge_id2,
        (const float*)d_aligned_pos1, (const float*)d_aligned_pos2,
        d_edge_indices1, d_edge_indices2, d_edge_id1, d_edge_id2);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // DtoH transfers
    gpuErrchk(hipMemcpy(&n_edge, d_n_edge, sizeof(int), hipMemcpyDeviceToHost));
    
    // Clamp n_edge to max_n_edge to prevent buffer overrun
    if (n_edge > max_n_edge) {
        n_edge = max_n_edge;
    }
    
    if (n_edge > 0) {
        gpuErrchk(hipMemcpy(edge_indices1, d_edge_indices1, n_edge * sizeof(int32_t), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(edge_indices2, d_edge_indices2, n_edge * sizeof(int32_t), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(edge_id1,      d_edge_id1,      n_edge * sizeof(int32_t), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(edge_id2,      d_edge_id2,      n_edge * sizeof(int32_t), hipMemcpyDeviceToHost));
    }

    // Free memory
    hipFree(d_aligned_pos1);
    if (!symmetric) hipFree(d_aligned_pos2);
    hipFree(d_cache_edge_indices1);
    hipFree(d_cache_edge_indices2);
    hipFree(d_cache_edge_id1);
    hipFree(d_cache_edge_id2);
    hipFree(d_edge_indices1);
    hipFree(d_edge_indices2);
    hipFree(d_edge_id1);
    hipFree(d_edge_id2);
    hipFree(d_n_edge);
}

} // extern "C"