#include "device_buffer.h"
#include "vector_math.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

// CUDA error-checking macro
static void handle_cuda_error(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error in " << file << " at line " << line << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define CUDA_CHECK(err) (handle_cuda_error(err, __FILE__, __LINE__))

template<typename T, int Dim>
DeviceBuffer<T, Dim>::DeviceBuffer(const VecArrayStorage& host)
    : host_storage_(&host), device_ptr_(nullptr), pitch_bytes_(0), 
      host_is_dirty_(true), device_is_dirty_(false) {
    
    if (Dim == 1) {
        // 1D allocation using hipMalloc
        size_t size_bytes = host.n_elem * host.row_width * sizeof(T);
        CUDA_CHECK(hipMalloc(&device_ptr_, size_bytes));
        pitch_bytes_ = host.row_width * sizeof(T);
    } else if (Dim == 2) {
        // 2D allocation using hipMallocPitch
        size_t width_bytes = host.row_width * sizeof(T);
        size_t height = host.n_elem;
        CUDA_CHECK(hipMallocPitch(reinterpret_cast<void**>(&device_ptr_), &pitch_bytes_, width_bytes, height));
    } else {
        throw std::runtime_error("DeviceBuffer only supports Dim=1 or Dim=2");
    }
}

template<typename T, int Dim>
DeviceBuffer<T, Dim>::~DeviceBuffer() {
    if (device_ptr_) {
        hipFree(device_ptr_);
    }
}

template<typename T, int Dim>
DeviceBuffer<T, Dim>::DeviceBuffer(DeviceBuffer&& other) noexcept
    : host_storage_(other.host_storage_), 
      device_ptr_(other.device_ptr_), 
      pitch_bytes_(other.pitch_bytes_),
      host_is_dirty_(other.host_is_dirty_),
      device_is_dirty_(other.device_is_dirty_) {
    other.device_ptr_ = nullptr;
    other.pitch_bytes_ = 0;
    other.host_is_dirty_ = false;
    other.device_is_dirty_ = false;
}

template<typename T, int Dim>
DeviceBuffer<T, Dim>& DeviceBuffer<T, Dim>::operator=(DeviceBuffer&& other) noexcept {
    if (this != &other) {
        if (device_ptr_) {
            hipFree(device_ptr_);
        }
        host_storage_ = other.host_storage_;
        device_ptr_ = other.device_ptr_;
        pitch_bytes_ = other.pitch_bytes_;
        host_is_dirty_ = other.host_is_dirty_;
        device_is_dirty_ = other.device_is_dirty_;
        other.device_ptr_ = nullptr;
        other.pitch_bytes_ = 0;
        other.host_is_dirty_ = false;
        other.device_is_dirty_ = false;
    }
    return *this;
}

template<typename T, int Dim>
void DeviceBuffer<T, Dim>::copyToDevice() {
    if (Dim == 1) {
        size_t size_bytes = host_storage_->n_elem * host_storage_->row_width * sizeof(T);
        CUDA_CHECK(hipMemcpy(device_ptr_, host_storage_->x.get(), size_bytes, hipMemcpyHostToDevice));
    } else if (Dim == 2) {
        size_t width_bytes = host_storage_->row_width * sizeof(T);
        size_t height = host_storage_->n_elem;
        CUDA_CHECK(hipMemcpy2D(device_ptr_, pitch_bytes_,
                              host_storage_->x.get(), host_storage_->row_width * sizeof(T),
                              width_bytes, height,
                              hipMemcpyHostToDevice));
    }
    host_is_dirty_ = false;
    device_is_dirty_ = false;  // After copy, both are in sync
}

template<typename T, int Dim>
void DeviceBuffer<T, Dim>::copyToHost() {
    if (Dim == 1) {
        size_t size_bytes = host_storage_->n_elem * host_storage_->row_width * sizeof(T);
        CUDA_CHECK(hipMemcpy(host_storage_->x.get(), device_ptr_, size_bytes, hipMemcpyDeviceToHost));
    } else if (Dim == 2) {
        size_t width_bytes = host_storage_->row_width * sizeof(T);
        size_t height = host_storage_->n_elem;
        CUDA_CHECK(hipMemcpy2D(host_storage_->x.get(), host_storage_->row_width * sizeof(T),
                              device_ptr_, pitch_bytes_,
                              width_bytes, height,
                              hipMemcpyDeviceToHost));
    }
    host_is_dirty_ = false;
    device_is_dirty_ = false;  // After copy, both are in sync
}

template<typename T, int Dim>
const T* DeviceBuffer<T, Dim>::devicePtr() const noexcept {
    return device_ptr_;
}

template<typename T, int Dim>
size_t DeviceBuffer<T, Dim>::pitch() const noexcept {
    return pitch_bytes_;
}

// Smart synchronization methods - host side
template<typename T, int Dim>
const VecArrayStorage* DeviceBuffer<T, Dim>::h_ptr() const {
    if (device_is_dirty_) {
        // Device has newer data, sync to host
        const_cast<DeviceBuffer*>(this)->copyToHost();
    }
    return host_storage_;
}

template<typename T, int Dim>
VecArrayStorage* DeviceBuffer<T, Dim>::h_ptr() {
    // First ensure we have the latest data
    const_cast<const DeviceBuffer*>(this)->h_ptr();
    // Mark host as dirty since caller can modify it
    host_is_dirty_ = true;
    device_is_dirty_ = false;
    return const_cast<VecArrayStorage*>(host_storage_);
}

// Smart synchronization methods - device side
template<typename T, int Dim>
const T* DeviceBuffer<T, Dim>::d_ptr() const {
    if (host_is_dirty_) {
        // Host has newer data, sync to device
        const_cast<DeviceBuffer*>(this)->copyToDevice();
    }
    return device_ptr_;
}

template<typename T, int Dim>
T* DeviceBuffer<T, Dim>::d_ptr() {
    // First ensure we have the latest data
    const_cast<const DeviceBuffer*>(this)->d_ptr();
    // Mark device as dirty since caller can modify it
    device_is_dirty_ = true;
    host_is_dirty_ = false;
    return device_ptr_;
}

// Explicit template instantiations for the types we need
template class DeviceBuffer<float, 1>;
template class DeviceBuffer<float, 2>;
